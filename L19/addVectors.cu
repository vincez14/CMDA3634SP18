#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"

// device function
__global__ void kernelAddVectors(double *a, double *b, double *c) {

	int threadid = threadIdx.x; // thread number
	int blockid = blockIdx.x; // block number
	int Nblock = blockDim.x; // number of threads in a block

	int id = threadid + blockid*Nblock;
    
    if (id < N) {
        double c = a[id] + b[id]; // this is private to this thread and on REGISTER
    
        for (int n=0; n<10; n++) {
            c *= c;
        }
        c[id] = c;
    }
}


int main(int argc, char **argv) {
	
    // get vector size from command line argument
	int N = atoi(argv[1]);

	// seed RNG
	double seed = clock();
	srand48(seed);

	double *h_a, *h_b, *h_c; // host vectors

	// allocate storage
	h_a = (double *) malloc(N*sizeof(double));
	h_b = (double *) malloc(N*sizeof(double));
	h_c = (double *) malloc(N*sizeof(double));

	// populate a and b
	for (int n =0; n<N; n++) {
		h_a[n] = drand48();
		h_b[n] = drand48();
	}

	double hostStart = clock();
	
	// c = a + b
	for (int n=0; n<N; n++) {
		h_c[n] = h_a[n] + h_b[n];
	}

	double hostEnd = clock();
	double hostTime = (hostEnd - hostStart)/(double) CLOCKS_PER_SEC;

	printf("The host took %f seconds to add a and b \n", hostTime);

	// device arrays
	double *d_a, *d_a, *d_c;

	hipMalloc(&d_a, N*sizeof(double));
	hipMalloc(&d_b, N*sizeof(double));
	hipMalloc(&d_c, N*sizeof(double));

	// copy data from the host to the device
	hipMemcpy(d_a, h_a, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(double), hipMemcpyHostToDevice);

    double copyEnd = clock();
    double copyTime = (copyEnd - copyStart)/(double) CLOCKS_PER_SEC;

    printf("It took %f seconds to copy the ", copyTime);

	// at this point the data is allocated and populated on the device
	int Nthreads = atoi(argv[2]); // get the number of threads per block from command line
	int Nblocks = (N+Nthreads-1)/Nthreads;

    double deviceStart = clock();

	kernelAddVectors <<<Nblocks, Nthreads >>>(d_a, d_b, d_c);

    hipDeviceSynchronize();

    double deviceEnd = clock();
    double deviceTime = (deviceEnd - deviceStart)/(double) CLOCKS_PER_SEC; 

    // free the variables after use
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
