#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

// safely compute a*b mod p
__device__ unsigned int modProdCuda(unsigned int a, unsigned int b, unsigned int p) {

  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab + za) % p;
    za = (2 * za) % p;
    b /= 2;
  }

  return ab;
}

// safely compute a^b mod p
__device__ unsigned int modExpCuda(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modProdCuda(aExpb, z, p);
    z = modProdCuda(z, z, p);
    b /= 2;
  }

  return aExpb;
}

// device function
__global__ void kernelDecrypt(unsigned int p, unsigned int g, unsigned int h, unsigned int x, unsigned int *d_r) {
  int threadid = threadIdx.x; // thread number
  int blockid = blockIdx.x; // block number
  int Nblock = blockDim.x; // number of threads in a block
  
  int id = threadid + blockid * Nblock;

  // find the secret key in parallel
  if (modExpCuda(g, id, p) == h) {
    *d_r = id;
  }

} // end kernelDecrypt()

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  FILE * pkfile, * msgfile;
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  char line[256];

  pkfile = fopen("bonus_public_key.txt", "r");
  msgfile = fopen("bonus_message.txt", "r");
 
  // parse through the public_key file and extract the information
  n = atoi(fgets(line, sizeof(line), pkfile));
  p = atoi(fgets(line, sizeof(line), pkfile));
  g = atoi(fgets(line, sizeof(line), pkfile));
  h = atoi(fgets(line, sizeof(line), pkfile));

  // close the public_key file
  fclose(pkfile);

  // status update
  printf("Retrieving cyphertext pairs.\n");

  // parse through the message file and extract the cyphertexts
  Nints = atoi(fgets(line, sizeof(line), msgfile));

  unsigned int charsPerInt = (n-1) / 8;
  unsigned int Nchars = charsPerInt * Nints;
  unsigned int *z = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));

  for (int i = 0; i < Nints; i++) {
    fgets(line, sizeof(line), msgfile);
    sscanf(line, "%u%u", &z[i], &a[i]);
  }

  // close the message file
  fclose(msgfile);

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();

    unsigned int numProc = p - 1;
    unsigned int *d_r; // storage for the device result

    // allocate memory on the device
    hipMalloc(&d_r, sizeof(unsigned int));

    int Nthreads = 32;
    int Nblocks = (numProc + Nthreads - 1) / Nthreads;

    // execute the kernel on the Device
    kernelDecrypt <<< Nblocks, Nthreads >>> (p, g, h, x, d_r);
    hipDeviceSynchronize();
    
    // copy the result over to the host
    hipMemcpy(&x, d_r, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    // FREE THE MEMORY
    hipFree(d_r);

    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  } // end finding the secret key

  /* Q3 After finding the secret key, decrypt the message */

  ElGamalDecrypt(z, a, Nints, p, x);
  convertZToString(z, Nints, message, Nchars);

  printf("Decrypted Message = \"%s\"\n", message);

  return 0;
}
