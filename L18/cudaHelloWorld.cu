#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

// this is my DEVICE function
// __global__ means this function is visible to the host
__global__ void kernelHelloWorld() {

	int thread = threadIdx.x; // local thread number in a block
	int block = blockIdx.x;	  // block number

	printf("Hello World from thread %d of block %d!\n", thread, block);

}

int main(int argc, char** argv) {

	int Nblocks = 10; // number of blocks
	int Nthreads = 3; // number of threads per block

	// run the function 'kernelHelloWorld' on the DEVICE
	kernelHelloWorld <<< Nblocks, Nthreads >>> ();

	// wait for the DEVICE function to complete before moving on
	hipDeviceSynchronize();

	return 0;	
}
